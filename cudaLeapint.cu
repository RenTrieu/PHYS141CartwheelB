#include "hip/hip_runtime.h"
/*
 * CUDALEAPINT.CU: program to integrate hamiltonian system using leapfrog 
 *                 and CUDA
 */

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "cudaLeapint.cuh"

int main(int argc, char **argv)
{
    /* Declaring Variables */
    int n, mstep, nout, nstep;
    double* rx = NULL;
    double* ry = NULL;
    double* rz = NULL;
    double* vx = NULL;
    double* vy = NULL;
    double* vz = NULL;

    // double* rx, ry, rz, vx, vy, vz;
    double tnow, dt;
    double nMass;

    /* GM Constants in AU^3/Day^2 */
    double GMCONST[MAXPNT];
    GMCONST[0] = 2.959E-4;
    for (int i = 1; i < MAXPNT; i++) {
        GMCONST[i] = 0.0;
    }

    /* Setting up Initial Conditions */

    /* Number of astronomical bodies */
    if (argc <= 1) {
        n = 10.0;
        nMass = 0.0;
    }
    else if (argc <= 2) {
        n = (int) atoi(argv[1]);
        nMass = 0.0;
    }
    else {
        n = (int) atoi(argv[1]);
        nMass = (double) atof(argv[2]);
    }

    /* Setting the gravitational constant for the peripheral bodies */
    for (int i = 1; i < MAXPNT; i++) {
        GMCONST[i] = nMass;
    }

    printf("Simulating %d particles with peripheral mass of %f\n", n, nMass);

    /* Setting initial time */
    tnow = 0.0;

    /* Allocating Unified Memory - accessible from CPU or GPU */
    hipMallocManaged(&rx, n*sizeof(double));
    hipMallocManaged(&ry, n*sizeof(double));
    hipMallocManaged(&rz, n*sizeof(double));
    hipMallocManaged(&vx, n*sizeof(double));
    hipMallocManaged(&vy, n*sizeof(double));
    hipMallocManaged(&vz, n*sizeof(double));

    /* Initializing Saturn */
    rx[0] = 0.0;					/* set initial x position */
    ry[0] = 0.0;                    /* set initial y position */
    rz[0] = 0.0;                    /* set initial z position */
    vx[0] = 0.0;					/* set initial x velocity */
    vy[0] = 0.0;                    /* set initial y velocity */
    vz[0] = 0.0;                    /* set initial z velocity */

    /* Determining equidistant angles */
    double nRadius = 0.001885*scaleFactor;           /* in AU * scaleFactor */
    double nAng = 360.0 / ((double) (n-1));
    double curAngle = nAng;
    for (int i = 1; i < n; i++) {
        rx[i] = nRadius*cos(curAngle*M_PI/180.0);
        ry[i] = nRadius*sin(curAngle*M_PI/180.0);
        rz[i] = 0.0;
        vx[i] = sqrt(GMCONST[0]/nRadius)*sin(curAngle*M_PI/180.0);
        vy[i] = -sqrt(GMCONST[0]/nRadius)*cos(curAngle*M_PI/180.0);
        vz[i] = 0.0;
        curAngle += nAng;
    }

    /* next, set integration parameters */

    mstep = 800;                     /* number of steps to take  */
    nout = 4;                        /* steps between outputs    */
    dt = 1.0;                        /* timestep for integration */

    /* now, loop performing integration */
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    /* Harmonic Oscillator */
    {
        FILE* outFile;
        const char * filename;
        const char * accelFormula;
        accelFormula = "solGravity";
        char accelTemp[strlen(accelFormula)+1];
        for (int i = 0; i < strlen(accelFormula); i++) {
            accelTemp[i] = accelFormula[i];
        }
        accelTemp[strlen(accelFormula)] = '\0';
        filename = strcat(accelTemp, "3D.txt");
        outFile = fopen(filename, "w");

        leapstep<<<numBlocks, blockSize>>>(rx, ry, rz,
                                           vx, vy, vz,
                                           n, dt, GMCONST);

        for (nstep = 0; nstep < mstep; nstep++) {	
            /* loop mstep times in all  */
            if (nstep % nout == 0) {
                /* if time to output state  */
                printstate(rx, ry, rz, 
                           vx, vy, vz, n, tnow, outFile, filename);
            }
            /* then call output routine */
            leapstep<<<numBlocks, blockSize>>>(rx, ry, rz,
                                          vx, vy, vz,
                                          n, dt, GMCONST);
            // leapstep(rx, ry, rz, vx, vy, vz, n, dt, accelFormula, GMCONST); 
            /* take integration step    */
            tnow = tnow + dt;			
            /* and update value of time */
        }
        if (mstep % nout == 0) {
            /* if last output wanted    */
            printstate(rx, ry, rz, 
                       vx, vy, vz, n, tnow, outFile, filename);
            /* then output last step    */
        }

        /* Outputting to file */
        printstate(rx, ry, rz, 
                   vx, vy, vz, n, tnow, outFile, filename);

    }
}

/*
 * LEAPSTEP: take one step using the leap-from integrator, formulated
 * as a mapping from t to t + dt.  WARNING: this integrator is not
 * accurate unless the timestep dt is fixed from one call to another.
 */

__global__
void leapstep(double rx[], double ry[], double rz[], 
              double vx[], double vy[], double vz[], 
              int n, double dt, double gmConst[])
{
    int i;
    double* ax;
    double* ay;
    double* az;
    
    ax = (double*) malloc(n*sizeof(double));
    ay = (double*) malloc(n*sizeof(double));
    az = (double*) malloc(n*sizeof(double));

    /* Acting acceleration on position */
    for (int i = 0; i < n; i++) {
        ax[i] = ay[i] = az[i] = 0;
        for (int j = 0; j < n; j++) {
            if (j != i) {
                double distVal = sqrt(((rx[i]-rx[j])*(rx[i]-rx[j]))
                                +((ry[i]-ry[j])*(ry[i]-ry[j]))
                                +((rz[i]-rz[j])*(rz[i]-rz[j])));
                if (distVal > 0.0) {
                    distVal = fabs(1/(distVal*distVal*distVal));
                }
                else {
                    distVal = 0.0;
                }
                ax[i] += -rx[i]*gmConst[j]*distVal;
                ay[i] += -ry[i]*gmConst[j]*distVal;
                az[i] += -rz[i]*gmConst[j]*distVal;
            }
        }
    }
    for (i = 0; i < n; i++) {
        /* loop over all points...  */
        vx[i] = vx[i] + 0.5 * dt * ax[i];
        vy[i] = vy[i] + 0.5 * dt * ay[i];
        vz[i] = vz[i] + 0.5 * dt * az[i];
        /* advance vel by half-step */
    }
    for (i = 0; i < n; i++) {
        /* loop over points again...*/
	    rx[i] = rx[i] + dt * vx[i];
	    ry[i] = ry[i] + dt * vy[i];
	    rz[i] = rz[i] + dt * vz[i];
        /* advance pos by full-step */
    }
    /* Acting acceleration on position */
    for (int i = 0; i < n; i++) {
        ax[i] = ay[i] = az[i] = 0;
        for (int j = 0; j < n; j++) {
            if (j != i) {
                double distVal = sqrt(((rx[i]-rx[j])*(rx[i]-rx[j]))
                                +((ry[i]-ry[j])*(ry[i]-ry[j]))
                                +((rz[i]-rz[j])*(rz[i]-rz[j])));
                if (distVal > 0.0) {
                    distVal = fabs(1/(distVal*distVal*distVal));
                }
                else {
                    distVal = 0.0;
                }
                ax[i] += -rx[i]*gmConst[j]*distVal;
                ay[i] += -ry[i]*gmConst[j]*distVal;
                az[i] += -rz[i]*gmConst[j]*distVal;
            }
        }
    }
    for (i = 0; i < n; i++) { 
        /* loop over all points...  */
	    vx[i] = vx[i] + 0.5 * dt * ax[i];
	    vy[i] = vy[i] + 0.5 * dt * ay[i];
	    vz[i] = vz[i] + 0.5 * dt * az[i];
        /* and complete vel. step   */
    }
}

/*
 * ACCEL: compute accelerations for harmonic oscillator(s).
 */

__global__
void accel(double* ax, double* ay, double* az, 
           double* rx, double* ry, double* rz, 
           int n, double gmConst[])
{
    /* Acting acceleration on position */
    for (int i = 0; i < n; i++) {
        ax[i] = ay[i] = az[i] = 0;
        for (int j = 0; j < n; j++) {
            if (j != i) {
                double distVal = sqrt(((rx[i]-rx[j])*(rx[i]-rx[j]))
                                +((ry[i]-ry[j])*(ry[i]-ry[j]))
                                +((rz[i]-rz[j])*(rz[i]-rz[j])));
                if (distVal > 0.0) {
                    distVal = fabs(1/(distVal*distVal*distVal));
                }
                else {
                    distVal = 0.0;
                }
                ax[i] += -rx[i]*gmConst[j]*distVal;
                ay[i] += -ry[i]*gmConst[j]*distVal;
                az[i] += -rz[i]*gmConst[j]*distVal;
            }
        }
    }
}

/*
 * PRINTSTATE: output system state variables.
 */

void printstate(double rx[], double ry[], double rz[],
                double vx[], double vy[], double vz[], 
                int n, double tnow, FILE* outFile, const char * filename)
{
    int i;
    outFile = fopen(filename, "a+");
    for (i = 0; i < n; i++)	{		
        /* loop over all points...  */
        fprintf(outFile, 
                "%8.4f\t%4d\t%20.6f\t%20.6f\t%20.6f\t%20.6f\t%20.6f\t%20.6f\n", 
                tnow, i, rx[i], ry[i], rz[i], vx[i], vy[i], vz[i]);
    }
    fclose(outFile);
}


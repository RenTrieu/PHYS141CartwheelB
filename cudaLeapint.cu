#include "hip/hip_runtime.h"
/*
 * CUDALEAPINT.CU: program to integrate hamiltonian system using leapfrog 
 *                 and CUDA
 */

#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "cudaLeapint.cuh"

int main(int argc, char **argv)
{
    /* Declaring Variables */
    int n, mstep, nout, nstep;
    float massBuffer[MAXPNT];
    float rxBuffer[MAXPNT];
    float ryBuffer[MAXPNT];
    float rzBuffer[MAXPNT];
    float vxBuffer[MAXPNT];
    float vyBuffer[MAXPNT];
    float vzBuffer[MAXPNT];
    float* mass = NULL;
    float* rx = NULL;
    float* ry = NULL;
    float* rz = NULL;
    float* vx = NULL;
    float* vy = NULL;
    float* vz = NULL;
    float* gm = NULL;
    char * nBodyFile;

    float tnow, dt;

    /* GM Constants in AU^3/Day^2 */
    float GMCONST[MAXPNT];
    for (int i = 0; i < MAXPNT; i++) {
        GMCONST[i] = 0.0;
    }

    /* Number of astronomical bodies */
    if (argc <= 1) {
        printf("Usage: %s [N-Body File]\n", argv[0]);
    }
    else {
        nBodyFile = argv[1];
    }

    /* Parsing through the nBodyFile for celestial bodies */
    printf("Reading values in from %s.\n", nBodyFile);
    FILE* fp = fopen((const char *) nBodyFile, "r");

    if (fp == NULL) {
        return 0;
    }

    int lineNumber = 0;
    char buffer[MAXBUFFER];
    char * delimiters = " \t";
    char * token;
    char * s;

    while (fgets(buffer, MAXBUFFER, fp)) {
        s = buffer;
        token = strtok(s, delimiters);
        if (token != NULL) {
            massBuffer[lineNumber] = atof(token);
        }
        token = strtok(NULL, delimiters);
        if (token != NULL) {
            rxBuffer[lineNumber] = atof(token);
        }
        token = strtok(NULL, delimiters);
        if (token != NULL) {
            ryBuffer[lineNumber] = atof(token);
        }
        token = strtok(NULL, delimiters);
        if (token != NULL) {
            rzBuffer[lineNumber] = atof(token);
        }
        token = strtok(NULL, delimiters);
        if (token != NULL) {
            vxBuffer[lineNumber] = atof(token);
        }
        token = strtok(NULL, delimiters);
        if (token != NULL) {
            vyBuffer[lineNumber] = atof(token);
        }
        token = strtok(NULL, delimiters);
        if (token != NULL) {
            vzBuffer[lineNumber] = atof(token);
        }
        lineNumber += 1;
    }
    n = lineNumber;
    fclose(fp);

    /* Allocating Unified Memory - accessible from CPU or GPU */
    hipSetDevice(0);
    hipMalloc(&mass, n*sizeof(float));
    hipMalloc(&rx, n*sizeof(float));
    hipMalloc(&ry, n*sizeof(float));
    hipMalloc(&rz, n*sizeof(float));
    hipMalloc(&vx, n*sizeof(float));
    hipMalloc(&vy, n*sizeof(float));
    hipMalloc(&vz, n*sizeof(float));
    hipMalloc(&gm, n*sizeof(float));

    /* Copying memory to the device */
    hipMemcpy(mass, massBuffer, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(rx, rxBuffer, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(ry, ryBuffer, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(rz, rzBuffer, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vx, vxBuffer, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vy, vyBuffer, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vz, vzBuffer, n*sizeof(float), hipMemcpyHostToDevice);


    /* Setting the gravitational constant for the peripheral bodies */
    for (int i = 0; i < n; i++) {
        GMCONST[i] = massBuffer[i];
    }
    hipMemcpy(gm, GMCONST, n*sizeof(float), hipMemcpyHostToDevice);


    /* Setting initial time */
    tnow = 0.0;

    /* next, set integration parameters */

    mstep = 500;                      /* number of steps to take  */
    nout = 1;                         /* steps between outputs    */
    dt = 0.02;                        /* timestep for integration */

    /* Checking to see if n is a multiple of nThreads*deviceCount 
       If not, then round down
       (As seen in David's code) */
    int nParticles = nThreads * int (float(n) / (nThreads));
    if (nParticles != n) {
        n = nParticles;
    }

    int numBlocks = n / nThreads;
    if (numBlocks == 0) {
        numBlocks = 1;
    }

    printf("numBlocks: %d\nnThreads: %d\n\nParticles: %d\n", numBlocks, nThreads, n);

    /* now, loop performing integration */

    /* Gravity Acceleration */
    {
        FILE* outFile;
        const char * filename;
        char * outBuffer = strtok(nBodyFile, ".");
        filename = strcat(outBuffer, "Sim.txt");
        outFile = fopen(filename, "w");

        /* Progress Bar Initialization */
        int maxBar = 30;
        char outputBar[maxBar];
        outputBar[0] = '|';
        outputBar[maxBar - 1]= '|';
        for (int i = 1; i < maxBar - 1; i++) {
            outputBar[i] = ' ';
        }

        printstate(rxBuffer, ryBuffer, rzBuffer, 
                   vxBuffer, vyBuffer, vzBuffer, n, tnow, outFile, filename);

        for (nstep = 0; nstep < mstep; nstep++) {	

            /* Progress Bar Handling/Management */
            int v = round((((double) nstep) / ((double) mstep)) * 100.0);
            int barIndex = (int) round(((double) v / 100.0) * maxBar) + 1;
            if (barIndex < maxBar - 1) {
                outputBar[barIndex] = '#';
            }
            printf("\r%s  %d%%", outputBar, v);
            fflush(stdout);

            /* then call output routine */

            leapstep <<<numBlocks, nThreads>>>(rx, ry, rz,
                                               vx, vy, vz,
                                               n, dt, gm, 0);
            hipDeviceSynchronize();

            /* Copying memory from device to computer */
            hipMemcpy(massBuffer, mass, n*sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(rxBuffer, rx, n*sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(ryBuffer, ry, n*sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(rzBuffer, rz, n*sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(vxBuffer, vx, n*sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(vyBuffer, vy, n*sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(vzBuffer, vz, n*sizeof(float), hipMemcpyDeviceToHost);

            /* take integration step    */
            tnow = tnow + dt;			
            /* and update value of time */

            /* Printing out current positions and velocities */
            if (nstep % nout == 0) {
                printstate(rxBuffer, ryBuffer, rzBuffer, 
                           vxBuffer, vyBuffer, vzBuffer, n, tnow, outFile, filename);
            }
        }
        hipDeviceSynchronize();

        /* Outputting to file */
        printstate(rxBuffer, ryBuffer, rzBuffer, 
                   vxBuffer, vyBuffer, vzBuffer, n, tnow, outFile, filename);
        /* Cleaning up progress bar */
        printf("\r%s  %d%%\n", outputBar, 100);
        fflush(stdout);
    }

    /* Freeing memory */
    hipFree(mass);
    hipFree(rx);
    hipFree(ry);
    hipFree(rz);
    hipFree(vx);
    hipFree(vy);
    hipFree(vz);
    hipFree(gm);
    hipDeviceReset();
}

/*
 * LEAPSTEP: take one step using the leap-from integrator, formulated
 * as a mapping from t to t + dt.  WARNING: this integrator is not
 * accurate unless the timestep dt is fixed from one call to another.
 */

__global__ void leapstep(float rx[], float ry[], float rz[], 
                         float vx[], float vy[], float vz[], 
                         int n, float dt, float gmConst[], int deviceOffset)
{
    int index = deviceOffset + blockIdx.x * blockDim.x + threadIdx.x;
    float3 ac3;

    /* call acceleration code */
    ac3 = accel(rx, ry, rz, n, gmConst, deviceOffset, index);
    __syncthreads();

    /* Applying acceleration to velocity */
    vx[index] = vx[index] + 0.5 * dt * ac3.x;
    vy[index] = vy[index] + 0.5 * dt * ac3.y;
    vz[index] = vz[index] + 0.5 * dt * ac3.z;

    /* Applying velocity to position */
    rx[index] = rx[index] + dt * vx[index];
    ry[index] = ry[index] + dt * vy[index];
    rz[index] = rz[index] + dt * vz[index];

    /* call acceleration code */
    ac3 = accel(rx, ry, rz, n, gmConst, deviceOffset, index);
    __syncthreads();

    vx[index] = vx[index] + 0.5 * dt * ac3.x;
    vy[index] = vy[index] + 0.5 * dt * ac3.y;
    vz[index] = vz[index] + 0.5 * dt * ac3.z;
}

/*
 * ACCEL: compute accelerations for harmonic oscillator(s).
 */

__device__
float3 accel(float* rx, float* ry, float* rz, 
             int n, float gmConst[], int deviceOffset, int index)
{
    float3 ac3 = {0.0f, 0.0f, 0.0f};
    if (index != 0) {
        for (int j = 0; j < n; j++) {
            if (j != index) {
                float distVal = (rx[index]-rx[j])*(rx[index]-rx[j])
                                +(ry[index]-ry[j])*(ry[index]-ry[j])
                                +(rz[index]-rz[j])*(rz[index]-rz[j])+0.00001;
                distVal = distVal * distVal * distVal;
                distVal = 1.0f / sqrtf(distVal);
        
                /* Summing up acceleration */
                ac3.x += -(rx[index]-rx[j])*gmConst[j]*distVal;
                ac3.y += -(ry[index]-ry[j])*gmConst[j]*distVal;
                ac3.z += -(rz[index]-rz[j])*gmConst[j]*distVal;
            }
            __syncthreads();
        }
    }
    return ac3;
}

/*
 * PRINTSTATE: output system state variables.
 */

void printstate(float rx[], float ry[], float rz[],
                float vx[], float vy[], float vz[], 
                int n, float tnow, FILE* outFile, const char * filename)
{
    int i;
    outFile = fopen(filename, "a+");
    for (i = 0; i < n; i++)	{		
        /* Printing out time, particle, position, and velocity */
        fprintf(outFile, 
                "%8.4f\t%4d\t%20.6f\t%20.6f\t%20.6f\t%20.6f\t%20.6f\t%20.6f\n", 
                tnow, i, rx[i], ry[i], rz[i], vx[i], vy[i], vz[i]);
    }
    fclose(outFile);
}

